#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


typedef struct City City;
extern "C" struct City{
    int totalPopulation;
    int density;

    int cityRanking;
    int lattitude;
    int longitude;
    char cityName[50];
    char state[2];

    struct City* connectedCitiesIndicies[];
    double edgeWeights[];
};


extern "C" struct InfectedCity{
    int susceptibleCount;
    int infectedCount;
    int recoveredCount;
    int iterationOfInfection;
};



extern "C" void covid_allocateMem( unsigned int** infectedCounts,
                        unsigned int** recoveredCounts,
                        unsigned int** infectedCountResults,
                        unsigned int** recoveredCountResults,
                        int numCities){

    int dataLength = numCities * sizeof(unsigned int);

    hipMallocManaged( infectedCounts, dataLength );
    hipMallocManaged( recoveredCounts, dataLength );
    hipMallocManaged( infectedCountResults, dataLength );
    hipMallocManaged( recoveredCountResults, dataLength );

}

extern "C" void gol_freeMem( unsigned int* infectedCounts,
                        unsigned int* recoveredCounts,
                        unsigned int* infectedCountResults,
                        unsigned int* recoveredCountResult){
    hipFree(infectedCounts);
    hipFree(recoveredCounts);
    hipFree(infectedCountResults);
    hipFree(recoveredCountResults);
}

static inline void pointer_swap( unsigned char **pA, unsigned char **pB)
{
    // You write this function - it should swap the pointers of pA and pB.
    //declare a temp to store A
    unsigned char * temp = *pA;
    //set a to b
    *pA = *pB;
    //set b to the stored val of a
    *pB = temp;
}


__global__ void covid_intracity_kernel(
                        City* cityData,
                        InfectedCity* allReleventInfectedCities,
                        InfectedCity* allReleventInfectedCitiesResult,
                        int dataLength)
{

    //parameters for SIR model
    double spreadRate = 2.2;
    double infectionDuration = 12.7;
    double recoveryRate = 1 / infectionDuration;

    //Declare variables that will be used
    int newInfections, newRecoveries;
    

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    while(index < dataLength){

        //get pointers to the indexed city
        City* city = allReleventInfectedCities + index;
        City* cityResult = allReleventInfectedCitiesResult + index;

        //SIR Model
        //new infections
        newInfections = (int) (spreadRate * city.susceptibleCount * city.infectedCount / cityData[index].totalPopulation);
        if(newInfections == 0 && city.susceptibleCount > 0) newInfections = 1;

        //new recoveries
        newRecoveries = (int) (recoveryRate * city.infectedCount);
        if(newRecoveries == 0 && city.susceptibleCount == 0 && city.infectedCount != 0) newRecoveries = 1;

        //Calculated city results
        cityResult.susceptibleCount = city.susceptibleCount - newInfections;
        cityResult.infectedCount    = city.infectedCount + newInfections - newRecoveries;
        cityResult.recoveredCount   = city.recoveredCount + newRecoveries;

        //increment the index
        index += blockDim.x * gridDim.x;

    }
    
}




__global__ void covid_spread_kernel(
                        City** cityData,
                        InfectedCity** allReleventInfectedCities,
                        InfectedCity** allReleventInfectedCitiesResult,
                        int dataLength)
{

    //Declare variables that will be used
    int infected, recovered;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    while(index < dataLength){

        //TODO

        //increment the index
        index += blockDim.x * gridDim.x;

    }
    
}



extern "C" bool covid_kernelLaunch( City** cityData,
                        InfectedCity** allReleventInfectedCities,
                        InfectedCity** allReleventInfectedCitiesResult,
                        int dataLength,
                        ushort threadsCount,
                        char intracity_or_spread)
{


    //calculate the number of blocks based on the threads per block
    int blockCount = dataLength / threadsCount;

    //run one itterations
    if(intracity_or_spread == 'i')
        covid_intracity_kernel<<<blockCount, threadsCount>>>( *cityData, *allReleventInfectedCities, *allReleventInfectedCitiesResult, dataLength);
    else if(intracity_or_spread == 's')
        covid_spread_kernel<<<blockCount, threadsCount>>>( *cityData, *allReleventInfectedCities, *allReleventInfectedCitiesResult, dataLength);

    pointer_swap( allReleventInfectedCities, allReleventInfectedCitiesResult );

    hipDeviceSynchronize();

    return true;
}







